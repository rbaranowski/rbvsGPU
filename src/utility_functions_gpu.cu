#include "utility_functions_gpu.cuh"


void pi_x_product_gpu(double *x, unsigned int n, unsigned int p, double *projection){
	
	double *d_x, *d_projection, *d_result;
	hipError_t err;

	hipMalloc( (void**)&d_x, n * p * sizeof(double));
	hipMalloc( (void**)&d_result, n * p * sizeof(double));
	hipMalloc( (void**)&d_projection, n * n * sizeof(double));

	hipMemcpy(d_x, x, n * p * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_projection, projection, n * n * sizeof(double), hipMemcpyHostToDevice);
   	
   	hipblasHandle_t handle;
	hipblasCreate(&handle);




	double alpha = 1.0;
	double beta = 0.0;

	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, p, n, &alpha, d_projection, n, d_x, n, &beta, d_result, n);
	
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("Cuda error: %s\n", hipGetErrorString(err));
	
	hipMemcpy(x, d_result, n * p * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_x);
	hipFree(d_result);
	hipFree(d_projection);
	hipblasDestroy(handle);


	err = hipGetLastError(); 
	if (err != hipSuccess) printf("Cuda error: %s\n", hipGetErrorString(err));


}
